#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <memory>
#include <vector>

#include <CUDA/HPP/Utils.hpp>

/**
 * # Initialization
 * 1.  hipdnnCreate
 * # Input
 * 2.  hipdnnCreateTensorDescriptor
 * 3.  hipdnnSetTensor4dDescriptor
 * # Filter
 * 4.  hipdnnCreateFilterDescriptor
 * 5.  hipdnnSetFilter4dDescriptor
 * # Convolution
 * 6.  hipdnnCreateConvolutionDescriptor
 * 7.  hipdnnSetConvolution2dDescriptor
 * # Output
 * 8.  hipdnnGetConvolution2dForwardOutputDim
 * 9.  hipdnnCreateTensorDescriptor
 * 10. hipdnnSetTensor4dDescriptor
 * # Algorithm
 * 11. hipdnnGetConvolutionForwardAlgorithm
 * 12. cudnnGetConvolutionBackwardAlgorithm
 * 13. hipdnnGetConvolutionBackwardFilterAlgorithm
 * # Workspace
 * 14. hipdnnGetConvolutionForwardWorkspaceSize
 * 15. hipdnnGetConvolutionBackwardDataWorkspaceSize
 * 16. hipdnnGetConvolutionBackwardFilterWorkspaceSize
 * # Forward propargation
 * 17. hipdnnConvolutionForward
 * # Backward propargation
 * 18. hipdnnConvolutionBackwardData
 * 19. hipdnnConvolutionBackwardFilter
 */

__global__
void InitData(float* px, float a, float b = -1.0) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (b != -1.0) {
    px[tid] = a;
  } else {
    px[tid] = tid + b;
  }
}

auto main() -> decltype(0) {

  const int xW = 5;
  const int xH = 5;
  const int xC = 1;
  const int xN = 1;

  const int wW = 2;
  const int wH = 2;
  const int wC = 1;
  const int wK = 1;

  const int padW = 0;
  const int padH = 0;

  const int strideW = 1;
  const int strideH = 1;

  const int dilationW = 1;
  const int dilationH = 1;

  const int xBias = 1;
  const int wBias = 1;

  // Initialization (creates context)
  hipdnnHandle_t ctx;
  cudnnCheck(hipdnnCreate(&ctx));

  // Input
  hipdnnTensorDescriptor_t xDesc;
  cudnnCheck(hipdnnCreateTensorDescriptor(&xDesc));
  cudnnCheck(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NCHW,
    HIPDNN_DATA_FLOAT, xN, xC, xH, xW));

  // Filter
  hipdnnFilterDescriptor_t wDesc;
  cudnnCheck(hipdnnCreateFilterDescriptor(&wDesc));
  cudnnCheck(hipdnnSetFilter4dDescriptor(wDesc, HIPDNN_DATA_FLOAT,
    HIPDNN_TENSOR_NCHW, wK, wC, wH, wW));

  // Convolution
  hipdnnConvolutionDescriptor_t cDesc;
  cudnnCheck(hipdnnCreateConvolutionDescriptor(&cDesc));
  cudnnCheck(hipdnnSetConvolution2dDescriptor(cDesc, padH, padW,
    strideH, strideW, dilationH, dilationW, HIPDNN_CONVOLUTION,
    HIPDNN_DATA_FLOAT));

  // Output
  int yN, yC, yH, yW;
  cudnnCheck(hipdnnGetConvolution2dForwardOutputDim(cDesc, xDesc, wDesc,
    &yN, &yC, &yH, &yW));

  hipdnnTensorDescriptor_t yDesc;
  cudnnCheck(hipdnnCreateTensorDescriptor(&yDesc));
  cudnnCheck(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW,
    HIPDNN_DATA_FLOAT, yN, yC, yH, yW));

  // Algorithm
  hipdnnConvolutionFwdAlgo_t fwdAlgo;
  hipdnnConvolutionBwdDataAlgo_t bwdXAlgo;
  hipdnnConvolutionBwdFilterAlgo_t bwdWAlgo;

  cudnnCheck(hipdnnGetConvolutionForwardAlgorithm(ctx, xDesc, wDesc, cDesc,
    yDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &fwdAlgo));
  cudnnCheck(hipdnnGetConvolutionBackwardDataAlgorithm(ctx, wDesc, yDesc,
    cDesc, xDesc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &bwdXAlgo));
  cudnnCheck(hipdnnGetConvolutionBackwardFilterAlgorithm(ctx, xDesc, yDesc,
    cDesc, wDesc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &bwdWAlgo));

  // Workspace
  std::size_t fwdWSsize;
  std::size_t bwdXWSsize;
  std::size_t bwdWWSsize;
  cudnnCheck(hipdnnGetConvolutionForwardWorkspaceSize(ctx, xDesc, wDesc, cDesc,
    yDesc, fwdAlgo, &fwdWSsize));
  cudnnCheck(hipdnnGetConvolutionBackwardDataWorkspaceSize(ctx, wDesc, yDesc,
    cDesc, xDesc, bwdXAlgo, &bwdXWSsize));
  cudnnCheck(hipdnnGetConvolutionBackwardFilterWorkspaceSize(ctx, xDesc, yDesc,
    cDesc, wDesc, bwdWAlgo, &bwdWWSsize));

  typedef float value_type;
  std::size_t value_size = sizeof(value_type);
  auto X = MemAlloc<value_type>(xN * xC * xH * xW * value_size);
  auto W = MemAlloc<value_type>(wK * wC * wH * wW * value_size);
  auto Y = MemAlloc<value_type>(yN * yC * yH * yW * value_size);
  auto dX = MemAlloc<value_type>(xN * xC * xH * xW * value_size);
  auto dW = MemAlloc<value_type>(wK * wC * wH * wW * value_size);
  auto dY = MemAlloc<value_type>(yN * yC * yH * yW * value_size);

  auto fwdWS = MemAlloc(fwdWSsize);
  auto bwdXWS = MemAlloc(bwdXWSsize);
  auto bwdWWS = MemAlloc(bwdWWSsize);

  InitData<<<xW * xH, xN * xC>>>(X.get(), xBias);
  InitData<<<wW * wH, wK * wC>>>(W.get(), wBias);
  InitData<<<xW * xH, xN * xC>>>(dX.get(), 0.0, 1.0);
  InitData<<<yW * yH, yN * yC>>>(dY.get(), 0.0, 0.0);
  InitData<<<wW * wH, wK * wC>>>(dW.get(), 0.0, 0.0);

  value_type alpha = 1.f;
  value_type beta = 0.f;
  cudnnCheck(hipdnnConvolutionForward(ctx, &alpha, xDesc, X.get(), wDesc,
    W.get(), cDesc, fwdAlgo, fwdWS.get(), fwdWSsize, &beta, yDesc, Y.get()));

  alpha = 1.f;
  beta = 1.f;
  cudnnCheck(hipdnnConvolutionBackwardData(ctx, &alpha, wDesc, W.get(), yDesc,
    dY.get(), cDesc, bwdXAlgo, bwdXWS.get(), bwdXWSsize, &beta, xDesc,
    dX.get()));

  cudnnCheck(hipdnnConvolutionBackwardFilter(ctx, &alpha, xDesc, X.get(), yDesc,
    dY.get(), cDesc, bwdWAlgo, bwdWWS.get(), bwdWWSsize, &beta, wDesc,
    dW.get()));

  cudnnCheck(hipdnnDestroyTensorDescriptor(xDesc));
  cudnnCheck(hipdnnDestroyTensorDescriptor(yDesc));
  cudnnCheck(hipdnnDestroyFilterDescriptor(wDesc));
  cudnnCheck(hipdnnDestroyConvolutionDescriptor(cDesc));
  cudnnCheck(hipdnnDestroy(ctx));

  return 0;
}
