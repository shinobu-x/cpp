#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include <hip/hip_runtime.h>
#include <>
#include <math_functions.h>

#include "../Hpp/CalElapsedTime.hpp"

__global__
void CalSine(const float* angle, float* sine, std::size_t size) {
  int thread = blockDim.x * blockIdx.x + threadIdx.x;
  if (thread < size) {
    sine[thread] = sinf(angle[thread]);
  }
}

void DoIt() {
  typedef float value_type;
  const std::size_t n = 100000000;
  std::size_t data_size = n * sizeof(float);

  // Hostでアドレス確保
  float* h_angle;
  float* h_sine;

  hipHostMalloc(&h_angle, data_size);
  hipHostMalloc(&h_sine, data_size);

  // Deviceでアドレス確保
  float* d_angle;
  float* d_sine;
  hipMalloc(&d_angle, data_size);
  hipMalloc(&d_sine, data_size);

  // データ転送: Host -> Device
  // hipMemcpyAsync(dest, src, size, type)
  hipMemcpyAsync(d_angle, h_angle, data_size, hipMemcpyDefault);
  // Kernel起動
  CalSine<<< (n + 255) / 256, 256 >>>(d_angle, d_sine, n);
  // データ転送: Device -> Host
  hipMemcpyAsync(h_angle, d_angle, data_size, hipMemcpyDefault);

  // 処理完了まで待機
  hipStreamSynchronize(nullptr);

  hipFree(d_angle);
  hipFree(d_sine);

  hipFree(h_angle);
  hipFree(h_sine);

  hipDeviceReset();
}

auto main() -> decltype(0) {
  std::cout << CalElapsedTime<>::Execution(DoIt) << '\n';
  return 0;
} 
