#include <cstdlib>
#include <hip/hip_runtime.h>

#include "../HPP/InitData.hpp"
#include "../HPP/cudaSetupDevice.hpp"
#include "../HPP/sumArraysOnDevice.hpp"


auto main() -> decltype(0) {
  cudaSetupDevice();

  typedef float value_type;
  int threads = 1 << 24;
  std::size_t nbytes = threads * sizeof(value_type);

  // Addresses reservations for Host
  value_type* h_a;
  value_type* h_b;
  value_type* h_c;
  h_a = (value_type*)malloc(nbytes);
  h_b = (value_type*)malloc(nbytes);
  h_c = (value_type*)malloc(nbytes);

  // Data initializations
  InitData(h_a, threads);
  InitData(h_b, threads);

  // Addresses reservations for Device
  value_type* d_a;
  value_type* d_b;
  value_type* d_c;
  hipMalloc((value_type**)&d_a, nbytes);
  hipMalloc((value_type**)&d_b, nbytes);
  hipMalloc((value_type**)&d_c, nbytes);

  // Copy data from host to device  
  hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, nbytes, hipMemcpyHostToDevice);

  // Execute kernel
  dim3 block(threads);
  dim3 grid((threads + block.x - 1) / block.x);
  sumArraysOnDevice<<<grid, block>>>(d_a, d_b, d_c, threads);
  hipDeviceSynchronize();

  // Copy data from device to host
  hipMemcpy(h_c, d_c, nbytes, hipMemcpyDeviceToHost);

  return 0;
}
