#include <cstdlib>
#include <hip/hip_runtime.h>
#include <CUDA/HPP/InitData.hpp>
#include <CUDA/HPP/sumArraysOnDevice.hpp>

template <typename ValueType>
void DoIt() {
  typedef ValueType value_type;
  int N = 1 << 24;
  std::size_t NBytes = N * sizeof(value_type);

  value_type* h_a;
  value_type* h_b;
  value_type* h_c;
  value_type* d_a;
  value_type* d_b;
  value_type* d_c;

  hipStream_t s1, s2;
  hipStreamCreate(&s1);
  hipStreamCreate(&s2);

  hipMalloc((void**)&d_a, NBytes);
  hipMalloc((void**)&d_b, NBytes);
  hipMalloc((void**)&d_c, NBytes);
  hipHostAlloc((void**)&h_a, NBytes, hipHostMallocDefault);
  hipHostAlloc((void**)&h_b, NBytes, hipHostMallocDefault);
  hipHostAlloc((void**)&h_c, NBytes, hipHostMallocDefault);

  InitData(h_a, N);
  InitData(h_b, N);

  hipMemcpyAsync(d_a, h_a, NBytes, hipMemcpyHostToDevice, s1);
  hipMemcpyAsync(d_b, h_b, NBytes, hipMemcpyHostToDevice, s2);

  dim3 Block(1024);
  dim3 Grid((N + Block.x - 1) / Block.x);
 
  sumArraysOnDevice<<<Grid, Block, 0, s1>>>(d_a, d_b, d_c, N);
  sumArraysOnDevice<<<Grid, Block, 0, s2>>>(d_a, d_b, d_c, N);

  hipMemcpyAsync(h_c, d_c, NBytes, hipMemcpyDeviceToHost, s1);
  hipMemcpyAsync(h_c, d_c, NBytes, hipMemcpyDeviceToHost, s2);

  hipDeviceSynchronize();

  hipFree(h_a);
  hipFree(h_b);
  hipFree(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

auto main() -> decltype(0) {
  DoIt<float>();

  return 0;
}
