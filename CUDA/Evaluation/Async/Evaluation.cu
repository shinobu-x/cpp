// nvprof --print-gpu-trace ./a.out
#include <boost/thread/future.hpp>
#include <boost/thread.hpp>
#include <cstdlib>
#include <CUDA/HPP/InitData.hpp>
#include <CUDA/HPP/sumArraysOnDevice.hpp>

typedef float value_type;

void SpawnKernel(int threads, std::size_t nbytes, hipStream_t stream) {
  value_type* h_a;
  value_type* h_b;
  value_type* h_c;
  h_a = (value_type*)malloc(nbytes);
  h_b = (value_type*)malloc(nbytes);
  h_c = (value_type*)malloc(nbytes);

  InitData(h_a, threads);
  InitData(h_b, threads);

  value_type* d_a;
  value_type* d_b;
  value_type* d_c;
  hipMalloc((value_type**)&d_a, nbytes);
  hipMalloc((value_type**)&d_b, nbytes);
  hipMalloc((value_type**)&d_c, nbytes);

  hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, nbytes, hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, nbytes, hipMemcpyHostToDevice);

  dim3 Block(threads);
  dim3 Grid((threads + Block.x - 1) / Block.x);
  sumArraysOnDevice<<<Grid, Block, 0, stream>>>(
    d_a, d_b, d_c, threads);
  hipMemcpyAsync(h_c, d_c, nbytes, hipMemcpyDeviceToHost, stream);
  hipDeviceSynchronize();

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

void EvaluateStream() {
  const int N = 8;
  hipStream_t s[N];
  value_type* Data[N];
  std::size_t NBytes = N * sizeof(value_type);

  for (int i = 0; i < N; ++i) {
    hipStreamCreate(&s[i]);
    hipMalloc(&Data[i], NBytes);
    SpawnKernel(N, NBytes, s[i]);
  }
}

auto main() -> decltype(0) {
  EvaluateStream();
  return 0;
}
