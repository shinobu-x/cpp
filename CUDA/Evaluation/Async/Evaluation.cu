#define BOOST_THREAD_VERSION 4
#include <boost/thread/future.hpp>
#include <boost/thread.hpp>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <CUDA/HPP/InitData.hpp>
#include <CUDA/HPP/sumArraysOnDevice.hpp>

typedef float value_type;
const int N = 8;
std::size_t NBytes = N * sizeof(value_type);

void SpawnKernel(hipStream_t stream = nullptr) {
  std::cout << boost::this_thread::get_id() << "\n";

  value_type* h_a;
  value_type* h_b;
  value_type* h_c;
  h_a = (value_type*)malloc(NBytes);
  h_b = (value_type*)malloc(NBytes);
  h_c = (value_type*)malloc(NBytes);

  InitData(h_a, N);
  InitData(h_b, N);

  value_type* d_a;
  value_type* d_b;
  value_type* d_c;
  hipMalloc(&d_a, NBytes);
  hipMalloc(&d_b, NBytes);
  hipMalloc(&d_c, NBytes);

  hipMemcpy(d_a, h_a, NBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, NBytes, hipMemcpyHostToDevice);

  dim3 Block(N);
  dim3 Grid((N + Block.x - 1) / Block.x);
  if (stream) {
    hipMemcpyAsync(d_a, h_a, NBytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, h_b, NBytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_c, h_c, NBytes, hipMemcpyHostToDevice, stream);
    sumArraysOnDevice<<<Grid, Block, 0, stream>>>(
      d_a, d_b, d_c, N);
    hipMemcpyAsync(h_c, d_c, NBytes, hipMemcpyDeviceToHost, stream);
  } else {
    hipMemcpy(d_a, h_a, NBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, NBytes, hipMemcpyHostToDevice);
    sumArraysOnDevice<<<Grid, Block>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
  }

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

void DoStream() {
  hipStream_t s[N];
  value_type* Data[N];

  for (int i = 0; i < N; ++i) {
    hipStreamCreate(&s[i]);
    hipMalloc(&Data[i], NBytes);
    SpawnKernel(s[i]);
  }

  for (int i = 0; i < N; ++i) {
    hipStreamSynchronize(s[i]);
    hipStreamDestroy(s[i]);
  }
}

void DoFuture() {
  boost::future<void> f[N];

  for (auto& f_ : f) {
    boost::packaged_task<void(hipStream_t)> t(SpawnKernel);
    f_ = t.get_future();
    boost::thread(boost::move(t), nullptr).detach();
  }

  for (auto& f_ : f) {
    f_.get();
    assert(f_.is_ready());
    assert(f_.has_value());
    assert(!f_.has_exception());
    assert(f_.get_state() == boost::future_state::ready);
  }
}

void DoAsync() {
  boost::future<void> f[N];

  for (int i = 0; i < N; ++i) {
    f[i] = boost::async(boost::launch::async, []() mutable {
      SpawnKernel(nullptr);
    });
  }

  for (auto& f_ : f) {
    f_.get();
    assert(f_.is_ready());
    assert(f_.has_value());
    assert(!f_.has_exception());
    assert(f_.get_state() == boost::future_state::ready);
  }

}

void Dummy() {
  std::cout << boost::this_thread::get_id() << "\n";
}

void Job1(boost::future<void> f) {
  std::cout << __func__ << ": Start!" << "\n";
  assert(f.valid());
  f.get();
  assert(!f.valid());
  SpawnKernel(nullptr);
  std::cout << __func__ << ": Done!" << "\n";
}

void Job2(boost::future<void> f) {
  std::cout << __func__ << ": Start!" << "\n";
  assert(f.valid());
  f.get();
  assert(!f.valid());
  SpawnKernel(nullptr);
  std::cout << __func__ << ": Done!" << "\n";
}

void Job3(boost::future<void> f) {
  std::cout << __func__ << ": Start!" << "\n";
  SpawnKernel(nullptr);
  assert(f.valid());
  f.get();
  assert(!f.valid());
  std::cout << __func__ << ": Done!" << "\n";
}

void Job4(boost::future<void> f) {
  std::cout << __func__ << ": Start!" << "\n";
  SpawnKernel(nullptr);
  assert(f.valid());
  f.get();
  assert(!f.valid());
  std::cout << __func__ << ": Done!" << "\n";
}

void Job5(boost::future<void> f) {
  std::cout << __func__ << ": Start!" << "\n";
  SpawnKernel(nullptr);
  assert(f.valid());
  f.get();
  assert(!f.valid());
  std::cout << __func__ << ": Done!" << "\n";
}

void DoContinuation() {
  boost::future<void> f1 = boost::async(boost::launch::async, &Dummy);
  assert(f1.valid());

  boost::future<void> f2 =
    f1.then(boost::launch::async, &Job1).
       then(boost::launch::async, &Job3).
       then(boost::launch::async, &Job5).
       then(boost::launch::async, &Job2).
       then(boost::launch::async, &Job4);
  assert(f2.valid());
  assert(!f1.valid());

  f2.get();
  assert(!f2.valid());
}

auto main() -> decltype(0) {
//  DoStream();
//  DoFuture();
//  DoAsync();
//  DoContinuation();
  return 0;
}
