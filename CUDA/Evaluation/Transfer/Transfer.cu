#include <cstdlib>
#include <hip/hip_runtime.h>
#include <CUDA/HPP/InitData.hpp>
#include <CUDA/HPP/sumArraysOnDevice.hpp>

typedef float value_type;
int N = 1 << 24;
std::size_t NBytes = N * sizeof(value_type);

value_type* h_a;
value_type* h_b;
value_type* h_c;
value_type* d_a;
value_type* d_b;
value_type* d_c;

template <typename T>
void DoKernel(T* a, T* b, T* c) {
  dim3 Block(1024);
  dim3 Grid((N + Block.x - 1) / Block.x);
  sumArraysOnDevice<<<Grid, Block>>>(a, b, c, N);
}

void Malloc() {
  h_a = (value_type*)malloc(NBytes);
  h_b = (value_type*)malloc(NBytes);
  h_c = (value_type*)malloc(NBytes);
  InitData(h_a, N);
  InitData(h_b, N);
  InitData(h_c, N);

  hipMalloc(&d_a, NBytes);
  hipMalloc(&d_b, NBytes);
  hipMalloc(&d_c, NBytes);

  hipMemcpy(d_a, h_a, NBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, NBytes, hipMemcpyHostToDevice);

  DoKernel(d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, NBytes, hipMemcpyDeviceToHost);
 
  hipStreamSynchronize(nullptr);
  
  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

void MallocHost() {
  hipHostMalloc(&h_a, NBytes);
  hipHostMalloc(&h_b, NBytes);
  hipHostMalloc(&h_c, NBytes);
  InitData(h_a, N);
  InitData(h_b, N);
  InitData(h_c, N);

  hipMalloc(&d_a, NBytes);
  hipMalloc(&d_b, NBytes);
  hipMalloc(&d_c, NBytes); 

  hipMemcpyAsync(d_a, h_a, NBytes, hipMemcpyDefault);
  hipMemcpyAsync(d_b, h_b, NBytes, hipMemcpyDefault);
  hipMemcpyAsync(d_c, h_c, NBytes, hipMemcpyDefault);

  DoKernel(d_a, d_b, d_c);

  hipMemcpyAsync(h_c, d_c, NBytes, hipMemcpyDefault);

  hipStreamSynchronize(nullptr);

  hipFree(h_a);
  hipFree(h_b);
  hipFree(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

auto main() -> decltype(0) {
//  Malloc();
//  MallocHost();
  return 0;
}
